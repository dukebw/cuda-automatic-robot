
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>

__global__ void hello_from_gpu(void)
{
        printf("Hello world from GPU, thread %d!\n", threadIdx.x);
}

int main(void)
{
        printf("Hello world from CPU!\n");
        hello_from_gpu<<<1, 10>>>();

        int32_t runtime_version;
        hipError_t cudaerr = hipRuntimeGetVersion(&runtime_version);
        assert(cudaerr == hipSuccess);

        int32_t driver_version;
        cudaerr = hipDriverGetVersion(&driver_version);
        assert(cudaerr == hipSuccess);

        printf("Runtime: %d, Driver: %d\n", runtime_version, driver_version);

        cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess)
                printf("Kernel launch failed with error %s\n",
                       hipGetErrorString(cudaerr));

        hipDeviceReset();

        return EXIT_SUCCESS;
}
