
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu(void)
{
        printf("Hello world from GPU, thread %d!\n", threadIdx.x);
}

int main(void)
{
        printf("Hello world from CPU!\n");
        hello_from_gpu<<<1, 10>>>();

        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess)
                printf("Kernel launch failed with error %s\n",
                       hipGetErrorString(cudaerr));

        hipDeviceReset();

        return EXIT_SUCCESS;
}
