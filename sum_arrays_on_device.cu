
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <assert.h>

__global__ void sum_arrays_on_device(float *c, float *a, float *b)
{
        uint32_t i = threadIdx.x + blockIdx.x*blockDim.x;
        c[i] = a[i] + b[i];

        /* printf("thread %d: %f + %f == %f\n", i, a[i], b[i], c[i]); */

        /* printf("threadIdx: (%d, %d, %d), blockIdx: (%d, %d, %d), " */
        /*        "blockDim: (%d, %d, %d), gridIdx: (%d, %d, %d)\n", */
        /*        threadIdx.x, threadIdx.y, threadIdx.z, */
        /*        blockIdx.x, blockIdx.y, blockIdx.z, */
        /*        blockDim.x, blockDim.y, blockDim.z, */
        /*        gridDim.x, gridDim.y, gridDim.z); */
}

void initialize_data(float *device_data, const uint32_t num_floats)
{
        hipError_t status;
        time_t t;
        uint32_t size_bytes = sizeof(float)*num_floats;
        float *temp_data = (float *)malloc(size_bytes);

        srand((uint32_t)time(&t));

        for (uint32_t i = 0;
             i < num_floats;
             ++i) {
                temp_data[i] = (float)(rand() & 0xFF)/10.0f;
        }

        status = hipMemcpy(device_data, temp_data, size_bytes, hipMemcpyHostToDevice);
        assert(status == hipSuccess);

        free(temp_data);
}

int main(void)
{
        hipError_t status;
        float *a;
        float *b;
        float *c;
        const uint32_t num_floats = 32;
        const uint32_t size_bytes = sizeof(float)*num_floats;

        status = hipMalloc(&a, size_bytes);
        assert(status == hipSuccess);

        status = hipMalloc(&b, size_bytes);
        assert(status == hipSuccess);

        status = hipMalloc(&c, size_bytes);
        assert(status == hipSuccess);

        initialize_data(a, num_floats);
        initialize_data(b, num_floats);

        dim3 block{num_floats/4};
        dim3 grid{(num_floats + (block.x - 1))/block.x};
        sum_arrays_on_device<<<grid, block>>>(c, a, b);

        status = hipDeviceSynchronize();
        assert(status == hipSuccess);

        status = hipFree(a);
        assert(status == hipSuccess);

        status = hipFree(b);
        assert(status == hipSuccess);

        status = hipFree(c);
        assert(status == hipSuccess);

        status = hipDeviceReset();
        assert(status == hipSuccess);

        return EXIT_SUCCESS;
}
